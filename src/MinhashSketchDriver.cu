#include "hip/hip_runtime.h"
// this is host-side app
// modified from MinhashSketch(https://github.com/daren996/MinhashSketch)

#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <list>
#include <cstdint>
#include <random>
#include <vector>
#include <string>

#include "MinhashSketch.cu"
#include "Hash.h"
#include "Utils.h"

using namespace std;

void output_signature(vector <vector<uint64>> sig1) {
    for (int h = 0; h < sig1.size(); ++h) {
        cout << "sig[" << h << "].size(): " << sig1[h].size() << "\t";
        for (int i = 0; i < sig1[h].size(); ++i) {
            cout << hex << sig1[h][i] << dec << " ";
        }
        cout << endl;
    }
}

void usage() {
    cout << "===========================" << endl;
    cerr << "Usage: " << endl << endl;
    cerr << "    ./MinhashSketch FILE_ONE FILE_TWO MODE" << endl;
    cerr << endl;
    cerr << "    Possible MODEs are:" << endl;
    cerr << endl << bold_on;
    cerr << "        all" << endl;
    cerr << endl;
    cerr << "        minhash_parallel" << endl;
    cerr << endl;
    cerr << "Execute \"MinhashSketch help\" for an extended help section." << endl;
    cout << "===========================" << endl;
    exit(1);
}

void help() {
    cout << endl;
    cout << bold_on << "NAME" << bold_off << endl;
    cout << "    " << "MinhashSketch" << endl;
    cout << endl;
    cout << bold_on << "USAGE" << bold_off << endl;
    cout << "    " << "MinhashSketch FILE_ONE FILE_TWO " << bold_on << "MODE [PARAMETERS...]" << bold_off << endl;
    cout << endl;
    cout << "    " << "MinhashSketch calculates the similarity between two text files FILE_ONE and FILE_TWO" << endl;
    cout << "    " << "and outputs it as a number between 0 and 1, where 1 means the two files are exactly" << endl;
    cout << "    " << "the same." << endl;
    cout << endl;
    cout << bold_on << "MODE" << bold_off << endl;
    cout << "    " << "There are modes which change the way MinhashSketch computes the similarity. " << endl;
    cout << "    " << "Each may make use of different parameters, indicated as follows:" << endl;;
    cout << endl;
    cout << "    " << bold_on << "all" << bold_off << endl;
    cout << "        " << "This option executes all modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "minhash" << bold_off << endl;
    cout << "        " << "Calculates the similarity by computing minhash signatures for each sequence. Used" << endl;
    cout << "        " << "parameters are." << endl;
    cout << endl;
    cout << "            " << "--k=POSITIVE_INTEGER as shingle size" << endl;
    cout << endl;
    cout << "            " << "--t=POSITIVE_INTEGER" << bold_on << " (obligatory) " << bold_off
         << "as number of hash functions used" << endl;
    cout << endl;
    cout << "            " << "--seed=INTEGER as random generator seed" << endl;
    cout << endl;
    cout << bold_on << "PARAMETERS" << bold_off << endl;
    cout << endl;
    cout << "    " << bold_on << "--k=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to k=9. Indicates the size of the shingles used to calculate the simi-" << endl;
    cout << "        " << "larity between the documents." << endl;
    cout << endl;
    cout << "    " << bold_on << "--m=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to m=1. Indicates the number of sketches saved in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--t=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to t=1. Indicates the number of hash functions used in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--seed=INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to a random value. Used by minhash modes in their random generator number." << endl;
    cout << endl;
    cout << "    " << bold_on << "-e" << bold_off << endl;
    cout << "        " << "Output in experimentation format." << endl;
    cout << endl;
    exit(0);
}

// Main function to run the program in mercator style
int run_mercator(const int k, const int m, const int t, char *dnaList, int length, uint64 *hashes_b, uint64_t* result)
{   

    MinhashSketch sketch;

    // number of blocks
    const int BLOCKS_NUM = sketch.getNBlocks();
    const int BLOCK_THREADS = 32 * 16;
    const int ITEMS_PER_THREAD = 4;
    cout << "Number of Blocks" << BLOCKS_NUM << "\n";
    int CHUNKS_NUM;
    int CHUNK_SIZE=BLOCKS_NUM * BLOCK_THREADS * ITEMS_PER_THREAD ;

    // Calculate how many chuncks does one have
    if (length % (BLOCKS_NUM * BLOCK_THREADS * ITEMS_PER_THREAD) == 0)
        CHUNKS_NUM = (length - k + 1) / CHUNK_SIZE;
    else
        CHUNKS_NUM = (length - k + 1) / CHUNK_SIZE + 1;

    for (int i = 0; i < (CHUNKS_NUM - 1); ++i) {
        // host_side buffers
        char *input = new char [CHUNK_SIZE];
        input = &dnaList + i*CHUNKS_SIZE ;
        uint64_t *output = new uint64_t [m];

        // begin MERCATOR usage
        Mercator::Buffer<char> ib(CHUNK_SIZE);
        Mercator::Buffer<uint64_t> ob(m);
        
        // move data into the input buffer
        ib.set(input, length);
        
            // pass in the program parameters
        sketch.getParams()->m=m;
        sketch.getParams()->k=k;
        sktech.BlockGetSketch.getParam()->thread_offset = """something_ add here""";
        sktech.BlockGetSketch.getParam()->hash_b = """something_ add here""";


        // set input and output place
        sketch.src.setSource(ib);
        sketch.snk.setSink(ob);
        
        sketch.run();
        ob.get(output, ob.size());
    
        //Print Results
    }

}

// MinhashSketch.exe ../testing_files/sequence_clip1.fasta ../testing_files/sequence_clip2.fasta all -e --k=5 --m=10 --t=10
int main(int argc, char *argv[]) {

    if (argc == 2 && string(argv[1]) == "help") help();
    if (argc < 4) usage();

    // DEFAULT VALUES
    int k, m, t, seed;
    bool e;
    k = 9;
    m = 1; // the number of sketches
    t = 1; // the number of hash functions
    seed = random_device()();
    e = false;

    // PARSE FILE_ONE FILE_TWO MODE
    string name_one = string(argv[1]);
    string name_two = string(argv[2]);
    string cal_name = string(argv[3]);
    ifstream file1(name_one);
    if (file1.fail()) {
        std::cerr << "Unable to open file 1" << name_one << std::endl;
        exit(1);
    }
    ifstream file2(name_two);
    if (file2.fail()) {
        std::cerr << "Unable to open file 2" << name_two << std::endl;
        exit(1);
    }

    // PARSE PARAMETERS
    for (int i = 4; i < argc; ++i) {
        string param(argv[i]);
        if (param == "-e") {
            e = true;
        } else {
            int param_size = (uint) param.size();
            if (param_size >= 5) {
                auto index_eq = (uint) param.find('=');
                if (index_eq + 2 <= param_size) {
                    string param_name = param.substr(0, index_eq);
                    if (param_name == "--k") {
                        k = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--m") {
                        m = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--t") {
                        t = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--seed") {
                        seed = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    }
                }
            }
        }
    }
    if (k < 1) {
        std::cerr << "K value too small! Minimum: 1" << std::endl;
        exit(1);
    }
    if (m < 1) {
        std::cerr << "M value too small! Minimum: 1" << std::endl;
        exit(1);
    }

    // GET TWO SEQUENCES
    string file_info1, file_info2, sequence1, sequence2, s1, s2;
    utils::file_to_string(file1, file_info1, sequence1); 
    utils::file_to_string(file2, file_info2, sequence2);
    uint64 sequence_size1 = sequence1.size(), sequence_size2 = sequence2.size();
    if (sequence1.size() < k || sequence2.size() < k) {
        cout << "k cannot be greater than the size of any document" << endl;
        exit(1);
    }
    file1.close();
    file2.close();
    cout << file_info1 << "\n" <<"sequence1 size: " << sequence1.size() << endl;
    cout << file_info2 << "\n" <<"sequence2 size: " << sequence2.size() << endl;
    // dnaLists are char array store the sequence in chars
    char dnaList1[sequence1.size()];
    char dnaList2[sequence2.size()];
    strcpy(dnaList1, sequence1.c_str());
    strcpy(dnaList2, sequence1.c_str());

    // MAIN PROGRESS
    clock_t ini_time;
    bool mode_found = false;
    double similarity, time;
    list <tuple<string, double, double>> results;

    //a list of hash randoms are calculated for t hash functions
    uint64 *hashes_b = generateHashes_b(t, seed);


    if (cal_name == "all" || cal_name == "minhash_parallel") {

        if (t < 1) {
            cerr << endl;
            cerr << "You must provide a parameter --t=POSITIVE_INTEGER parameter for minhash modes!" 
                    << endl << endl;
            exit(1);
        }

        mode_found = true;
        ini_time = clock();

        sig2  = new uint64_t[m];
        sig1  = new uint64_t[m];

        //
        // Changes should be made here to use Mercator version of codes
        run_mercator(k, m, t, dnaList1, sequence1.size(), hashes_b, sig1);
        run_mercator(k, m, t, dnaList2, sequence2.size(), hashes_b, sig2);
        
        // Change the original output type to be fixed size array
        // vector <vector<uint64>> sig1 = genSig(k, m, t, dnaList1, sequence1.size(), hashes_b);
        // vector <vector<uint64>> sig2 = genSig(k, m, t, dnaList2, sequence2.size(), hashes_b);


        cout << "sig1 size:" << sig1[0].size() << endl;
        output_signature(sig1);
        cout << "\nsig2 size:" << sig2[0].size() << endl;
        output_signature(sig2);
        cout << endl;
        similarity = computeSim(sig1, sig2);
        time = double(clock() - ini_time) / CLOCKS_PER_SEC;
        results.emplace_back("minhash_parallel", similarity, time);
    }
    if (!mode_found) usage();

    // OUTPUT RESULTS
    if (e) {
        cout << setw(12) << "cal_name" << setw(14) << "seed" << setw(9) << "k" << setw(5) << "m" << setw(7) << "t";
        cout << setw(9) << fixed << "time" << setw(13) << fixed << "similarity" << endl;
    } else {
        cout << "===========================" << endl;
        cout << "k:" << k << setw(7) << fixed << "m:" << m << setw(7) << fixed << "t:" << t << endl;
        cout << "===========================" << endl;
    }
    cout.precision(8);
    for (auto &result : results) {
        if (e) {
            cout << setw(12) << get<0>(result) << setw(14) << seed << setw(5) << k <<9setw(5) << m << setw(7) << t;
            cout << setw(9) << fixed << get<2>(result) << setw(13) << fixed << get<1>(result) << endl;
        } else {
            cout << uline_on << get<0>(result) << uline_off << endl;
            cout << "time: " << setw(21) << fixed << get<2>(result) << endl;
            cout << "similarity: " << setw(15) << fixed << get<1>(result) << endl;
            cout << "===========================" << endl;
        }
    }

}