#include "hip/hip_runtime.h"
// this is host-side app
// modified from MinhashSketch(https://github.com/daren996/MinhashSketch)

#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <list>
#include <cstdint>
#include <random>
#include <vector>
#include <string>

#include "MinhashSketch.cu"
#include "Hash.h"
#include "Utils.h"

using namespace std;

void usage() {
    cout << "===========================" << endl;
    cerr << "Usage: " << endl << endl;
    cerr << "    ./MinhashSketch FILE_ONE FILE_TWO MODE" << endl;
    cerr << endl;
    cerr << "    Possible MODEs are:" << endl;
    cerr << endl << bold_on;
    cerr << "        all" << endl;
    cerr << endl;
    cerr << "        minhash_parallel" << endl;
    cerr << endl;
    cerr << "Execute \"MinhashSketch help\" for an extended help section." << endl;
    cout << "===========================" << endl;
    exit(1);
}

void help() {
    cout << endl;
    cout << bold_on << "NAME" << bold_off << endl;
    cout << "    " << "MinhashSketch" << endl;
    cout << endl;
    cout << bold_on << "USAGE" << bold_off << endl;
    cout << "    " << "MinhashSketch FILE_ONE FILE_TWO " << bold_on << "MODE [PARAMETERS...]" << bold_off << endl;
    cout << endl;
    cout << "    " << "MinhashSketch calculates the similarity between two text files FILE_ONE and FILE_TWO" << endl;
    cout << "    " << "and outputs it as a number between 0 and 1, where 1 means the two files are exactly" << endl;
    cout << "    " << "the same." << endl;
    cout << endl;
    cout << bold_on << "MODE" << bold_off << endl;
    cout << "    " << "There are modes which change the way MinhashSketch computes the similarity. " << endl;
    cout << "    " << "Each may make use of different parameters, indicated as follows:" << endl;;
    cout << endl;
    cout << "    " << bold_on << "all" << bold_off << endl;
    cout << "        " << "This option executes all modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "minhash" << bold_off << endl;
    cout << "        " << "Calculates the similarity by computing minhash signatures for each sequence. Used" << endl;
    cout << "        " << "parameters are." << endl;
    cout << endl;
    cout << "            " << "--k=POSITIVE_INTEGER as shingle size" << endl;
    cout << endl;
    cout << "            " << "--t=POSITIVE_INTEGER" << bold_on << " (obligatory) " << bold_off
         << "as number of hash functions used" << endl;
    cout << endl;
    cout << "            " << "--seed=INTEGER as random generator seed" << endl;
    cout << endl;
    cout << bold_on << "PARAMETERS" << bold_off << endl;
    cout << endl;
    cout << "    " << bold_on << "--k=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to k=9. Indicates the size of the shingles used to calculate the simi-" << endl;
    cout << "        " << "larity between the documents." << endl;
    cout << endl;
    cout << "    " << bold_on << "--m=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to m=1. Indicates the number of sketches saved in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--t=POSITIVE_INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to t=1. Indicates the number of hash functions used in minhash modes." << endl;
    cout << endl;
    cout << "    " << bold_on << "--seed=INTEGER" << bold_off << endl;
    cout << "        " << "Defaults to a random value. Used by minhash modes in their random generator number." << endl;
    cout << endl;
    cout << "    " << bold_on << "-e" << bold_off << endl;
    cout << "        " << "Output in experimentation format." << endl;
    cout << endl;
    exit(0);
}

// Main function to run the program in mercator style
int run_mercator(const int k, const int m, const int t, char *dnaList, int length, uint64 hash_b, uint64_t result[])
{   

    MinhashSketch sketch;

    // 3 GPU parameters, used to decide the size of chunck 
    const int BLOCKS_NUM = sketch.getNBlocks();
    const int BLOCK_THREADS = 32 * 16;
    const int ITEMS_PER_THREAD = 1;
    cout << "GPU info:\n Number of Blocks: " << BLOCKS_NUM << "\n";
    cout << "BLOCK_THREADS: " << BLOCK_THREADS << "\n";
    cout << "ITEMS_PER_THREAD: " << ITEMS_PER_THREAD << "\n";

    // size of each chunk and number of chunks needed for this sequence 
    int CHUNKS_NUM;
    int CHUNK_SIZE=BLOCKS_NUM * BLOCK_THREADS * ITEMS_PER_THREAD;

    // Calculate how many chuncks does one have
    if (length % (CHUNK_SIZE) == 0)
        CHUNKS_NUM = (length - k + 1) / CHUNK_SIZE;
    else
        CHUNKS_NUM = (length - k + 1) / CHUNK_SIZE + 1;
k
    // each chuck is loaded into input every time
    char *input = new char * [CHUNK_SIZE];
    uint64_t *output = new uint64_t [m];
    uint64_t *output_dev;
    // assign sapce on device to hold result
    res = hipMalloc( (void**) &output_dev, sizeof(uint64_t)*m );
    CHECK(res);

    // calculate the first CHUNK_NUM -1 chuncks, the last obne might have a different size
    for (int i = 0; i < (CHUNKS_NUM - 1); i ++) {
        // host_side buffers
        
        // load the target sequence chunk into input var
        for(int j = 0; j < CHUNK_SIZE; j ++){
            input[j] = &dnaList[i*CHUNK_SIZE + j];
        }
        
        // begin MERCATOR usage
        Mercator::Buffer<char*> ib(CHUNK_SIZE);
        Mercator::Buffer<uint64_t> ob(m);
        
        // move data into the input buffer
        ib.set(input, CHUNK_SIZE);
        
        // pass in the program parameters
        sketch.getParams()->m=m;
        sketch.getParams()->k=k;
        sktech.BlockGetSketch.getParam()->hash_b = hash_b;
        sktech.MergeSketch.getParam()->resultSketchStorage = output_dev;

        // set input and output place
        sketch.src.setSource(ib);
        sketch.mergeSketch.setSink(ob);
        
        sketch.run();

        ob.get(output, ob.size());
    
        //Print Results
    }
    // deal with the last chunk
    int LAST_CHUNK_SIZE = length%CHUNK_SIZE + 1;
    int LAST_CHUNK_START = length-LAST_CHUNK_SIZE+1 ;
    char *input = new char [LAST_CHUNK_SIZE];
    for(int j = LAST_CHUNK_START ; j < length ; j ++){
        input[j- LAST_CHUNK_START] = dnaList[j];
    }


    return 0;
}

// MinhashSketch.exe ../testing_files/sequence_clip1.fasta ../testing_files/sequence_clip2.fasta all -e --k=5 --m=10 --t=10
int main(int argc, char *argv[]) {

    if (argc == 2 && string(argv[1]) == "help") help();
    if (argc < 4) usage();

    // DEFAULT VALUES
    int k, m, t, seed;
    bool e;
    k = 9;
    m = 1; // the number of sketches
    t = 1; // the number of hash functions
    seed = random_device()();
    e = false;

    // PARSE FILE_ONE FILE_TWO MODE
    string name_one = string(argv[1]);
    string name_two = string(argv[2]);
    string cal_name = string(argv[3]);
    ifstream file1(name_one);
    if (file1.fail()) {
        std::cerr << "Unable to open file 1" << name_one << std::endl;
        exit(1);
    }
    ifstream file2(name_two);
    if (file2.fail()) {
        std::cerr << "Unable to open file 2" << name_two << std::endl;
        exit(1);
    }

    // PARSE PARAMETERS
    for (int i = 4; i < argc; ++i) {
        string param(argv[i]);
        if (param == "-e") {
            e = true;
        } else {
            int param_size = (uint) param.size();
            if (param_size >= 5) {
                auto index_eq = (uint) param.find('=');
                if (index_eq + 2 <= param_size) {
                    string param_name = param.substr(0, index_eq);
                    if (param_name == "--k") {
                        k = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--m") {
                        m = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--t") {
                        t = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    } else if (param_name == "--seed") {
                        seed = std::stoi(param.substr(index_eq + 1, param_size - index_eq - 1));
                    }
                }
            }
        }
    }
    if (k < 1) {
        std::cerr << "K value too small! Minimum: 1" << std::endl;
        exit(1);
    }
    if (m < 1) {
        std::cerr << "M value too small! Minimum: 1" << std::endl;
        exit(1);
    }

    // GET TWO SEQUENCES
    string file_info1, file_info2, sequence1, sequence2, s1, s2;
    utils::file_to_string(file1, file_info1, sequence1); 
    utils::file_to_string(file2, file_info2, sequence2);
    uint64 sequence_size1 = sequence1.size(), sequence_size2 = sequence2.size();
    if (sequence1.size() < k || sequence2.size() < k) {
        cout << "k cannot be greater than the size of any document" << endl;
        exit(1);
    }
    file1.close();
    file2.close();
    cout << file_info1 << "\n" <<"sequence1 size: " << sequence1.size() << endl;
    cout << file_info2 << "\n" <<"sequence2 size: " << sequence2.size() << endl;
    // dnaLists are char array store the sequence in chars
    char dnaList1[sequence1.size()];
    char dnaList2[sequence2.size()];
    strcpy(dnaList1, sequence1.c_str());
    strcpy(dnaList2, sequence1.c_str());

    // MAIN PROGRESS
    clock_t ini_time;
    bool mode_found = false;
    double similarity, time;

    //a list of hash randoms are calculated for t hash functions
    // uint64 *hashes_b = generateHashes_b(t, seed);
    // change to use single hash_b
    uint64 hash_b = generateHashes_b(1, seed);

    if (cal_name == "all" || cal_name == "minhash_parallel") {

        // if (t < 1) {
        //     cerr << endl;
        //     cerr << "You must provide a parameter --t=POSITIVE_INTEGER parameter for minhash modes!" 
        //             << endl << endl;
        //     exit(1);
        // }

        mode_found = true;
        ini_time = clock();

        uint64_t *sig2  = new uint64_t[m];
        uint64_t *sig1  = new uint64_t[m];

        //
        // Changes should be made here to use Mercator version of codes
        run_mercator(k, m, t, dnaList1, sequence1.size(), hash_b, sig1);
        run_mercator(k, m, t, dnaList2, sequence2.size(), hash_b, sig2);
        
        // Change the original output type to be fixed size array
        // vector <vector<uint64>> sig1 = genSig(k, m, t, dnaList1, sequence1.size(), hashes_b);
        // vector <vector<uint64>> sig2 = genSig(k, m, t, dnaList2, sequence2.size(), hashes_b);

        // somputeSim now need to be change to take in 2 uint64 array
        similarity = computeSim(sig1, sig2);
        time = double(clock() - ini_time) / CLOCKS_PER_SEC;
        results.emplace_back("minhash_parallel", similarity, time);
    }
    if (!mode_found) usage();

    // OUTPUT RESULTS
    if (e) {
        cout << setw(12) << "cal_name" << setw(14) << "seed" << setw(9) << "k" << setw(5) << "m" << setw(7) << "t";
        cout << setw(9) << fixed << "time" << setw(13) << fixed << "similarity" << endl;
    } else {
        cout << "===========================" << endl;
        cout << "k:" << k << setw(7) << fixed << "m:" << m << setw(7) << fixed << "t:" << t << endl;
        cout << "===========================" << endl;
    }
    cout.precision(8);
    for (auto &result : results) {
        if (e) {
            cout << setw(12) << get<0>(result) << setw(14) << seed << setw(5) << k <<9setw(5) << m << setw(7) << t;
            cout << setw(9) << fixed << get<2>(result) << setw(13) << fixed << get<1>(result) << endl;
        } else {
            cout << uline_on << get<0>(result) << uline_off << endl;
            cout << "time: " << setw(21) << fixed << get<2>(result) << endl;
            cout << "similarity: " << setw(15) << fixed << get<1>(result) << endl;
            cout << "===========================" << endl;
        }
    }

}